
#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdlib>
#include <ctime>
#include <cassert>
#include <zlib.h>
#include <png.h>
#include <math.h>

#define num_cluster 1000
#define INF 1000000
#define BlockFactor 32
//#define HB 32

unsigned char *image_src, *image_result;
unsigned char *device_image_src, *device_image_result;

int read_png(const char* filename, unsigned char** image, unsigned* height, 
             unsigned* width, unsigned* channels) {

    unsigned char sig[8];
    FILE* infile;
    infile = fopen(filename, "rb");

    fread(sig, 1, 8, infile);
    if (!png_check_sig(sig, 8))
        return 1;   /* bad signature */

    png_structp png_ptr;
    png_infop info_ptr;

    png_ptr = png_create_read_struct(PNG_LIBPNG_VER_STRING, NULL, NULL, NULL);
    if (!png_ptr)
        return 4;   /* out of memory */
  
    info_ptr = png_create_info_struct(png_ptr);
    if (!info_ptr) {
        png_destroy_read_struct(&png_ptr, NULL, NULL);
        return 4;   /* out of memory */
    }

    png_init_io(png_ptr, infile);
    png_set_sig_bytes(png_ptr, 8);
    png_read_info(png_ptr, info_ptr);
    int bit_depth, color_type;
    png_get_IHDR(png_ptr, info_ptr, width, height, &bit_depth, &color_type, NULL, NULL, NULL);

    png_uint_32  i, rowbytes;
    png_bytep  row_pointers[*height];
    png_read_update_info(png_ptr, info_ptr);
    rowbytes = png_get_rowbytes(png_ptr, info_ptr);
    *channels = (int) png_get_channels(png_ptr, info_ptr);

    if ((*image = (unsigned char *) malloc(rowbytes * *height)) == NULL) {
        png_destroy_read_struct(&png_ptr, &info_ptr, NULL);
        return 3;
    }

    for (i = 0;  i < *height;  ++i)
        row_pointers[i] = *image + i * rowbytes;
    png_read_image(png_ptr, row_pointers);
    png_read_end(png_ptr, NULL);
    return 0;
}

void write_png(const char* filename, png_bytep image, const unsigned height, const unsigned width, 
               const unsigned channels) {
    FILE* fp = fopen(filename, "wb");
    png_structp png_ptr = png_create_write_struct(PNG_LIBPNG_VER_STRING, NULL, NULL, NULL);
    png_infop info_ptr = png_create_info_struct(png_ptr);
    png_init_io(png_ptr, fp);
    png_set_IHDR(png_ptr, info_ptr, width, height, 8,
                 PNG_COLOR_TYPE_RGB, PNG_INTERLACE_NONE,
                 PNG_COMPRESSION_TYPE_DEFAULT, PNG_FILTER_TYPE_DEFAULT);
    png_set_filter(png_ptr, 0, PNG_NO_FILTERS);
    png_write_info(png_ptr, info_ptr);
    png_set_compression_level(png_ptr, 1);

    png_bytep row_ptr[height];
    for (int i = 0; i < height; ++ i) {
        row_ptr[i] = image + i * width * channels * sizeof(unsigned char);
    }
    png_write_image(png_ptr, row_ptr);
    png_write_end(png_ptr, NULL);
    png_destroy_write_struct(&png_ptr, &info_ptr);
    fclose(fp);
}

inline __device__ int bound_check(int val, int lower, int upper) {
    if (val >= lower && val < upper)
        return 1;
    else
        return 0;
}

__global__ void KmeansAssignCluster(unsigned char* image_src, unsigned char* centroid, int* num_pt_cluster,\
                                     int* sum_dist, int height, int width, int channels) {

    const int idx_x = blockIdx.x * blockDim.x + threadIdx.x;
    const int idx_y = blockIdx.y * blockDim.y + threadIdx.y;
    const int thread_width = blockDim.x * gridDim.x;
    const int thread_height = blockDim.y * gridDim.y;

    __shared__ unsigned char shared_centroid[num_cluster][3];
    __shared__ int shared_sum_dist[num_cluster][3];
    __shared__ int shared_num_pt_cluster[num_cluster];

    int min_dist = INF;
    int dist = 0;
    int cluster_idx;

    unsigned char img_src0 = image_src[channels * (idx_x + idx_y * width) + 0];
    unsigned char img_src1 = image_src[channels * (idx_x + idx_y * width) + 1];
    unsigned char img_src2 = image_src[channels * (idx_x + idx_y * width) + 2];

    for(int k = (threadIdx.x + threadIdx.y * blockDim.x); k < num_cluster; k += (blockDim.x * blockDim.y)) {
        shared_centroid[k][0] = centroid[channels * k + 0];
        shared_centroid[k][1] = centroid[channels * k + 1];
        shared_centroid[k][2] = centroid[channels * k + 2];
        shared_num_pt_cluster[k] = 0;
        shared_sum_dist[k][0] = 0;
        shared_sum_dist[k][1] = 0;
        shared_sum_dist[k][2] = 0;
    }

   __syncthreads();

    if(!(bound_check(idx_x, 0, width) && bound_check(idx_y, 0, height))) {
        return;
    }

    for(int k = 0; k < num_cluster; k++) {
        // calculate l2 norm
        dist = 0;
        dist += (img_src0 - shared_centroid[k][0]) * (img_src0 - shared_centroid[k][0]);
        dist += (img_src1 - shared_centroid[k][1]) * (img_src1 - shared_centroid[k][1]);
        dist += (img_src2 - shared_centroid[k][2]) * (img_src2 - shared_centroid[k][2]);
        dist = sqrt((float)dist);

        if(dist < min_dist) {
            min_dist = dist;
            cluster_idx = k;
        }
    }

    atomicAdd(&shared_num_pt_cluster[cluster_idx], 1);
    atomicAdd(&shared_sum_dist[cluster_idx][0], img_src0);
    atomicAdd(&shared_sum_dist[cluster_idx][1], img_src1);
    atomicAdd(&shared_sum_dist[cluster_idx][2], img_src2);

    for(int k = (threadIdx.x + threadIdx.y * blockDim.x); k < num_cluster; k += (blockDim.x * blockDim.y)) {
        atomicAdd(&num_pt_cluster[k], shared_num_pt_cluster[k]);
        atomicAdd(&sum_dist[k * channels + 0], shared_sum_dist[k][0]);
        atomicAdd(&sum_dist[k * channels + 1], shared_sum_dist[k][0]);
        atomicAdd(&sum_dist[k * channels + 2], shared_sum_dist[k][0]);
    }

}

__global__ void KmeansCalLoss(unsigned char* new_centroid, unsigned char* centroid, int* sum_loss, int* sum_dist, int* num_pt_cluster, unsigned int channels) {

    const int cluster_idx = blockIdx.x * blockDim.x + threadIdx.x;

    int dist = 0;

    int new_centroid_0;// = new_centroid[channels * cluster_idx + 0];
    int new_centroid_1;// = new_centroid[channels * cluster_idx + 1];
    int new_centroid_2;// = new_centroid[channels * cluster_idx + 1];
    int centroid_0 = centroid[channels * cluster_idx + 0];
    int centroid_1 = centroid[channels * cluster_idx + 1];
    int centroid_2 = centroid[channels * cluster_idx + 1];

    new_centroid_0 = new_centroid[channels * cluster_idx + 0] = sum_dist[channels * cluster_idx + 0] / num_pt_cluster[cluster_idx];
    new_centroid_1 = new_centroid[channels * cluster_idx + 1] = sum_dist[channels * cluster_idx + 1] / num_pt_cluster[cluster_idx];
    new_centroid_2 = new_centroid[channels * cluster_idx + 2] = sum_dist[channels * cluster_idx + 2] / num_pt_cluster[cluster_idx];

    dist += (new_centroid_0 - centroid_0) * (new_centroid_0 - centroid_0);
    dist += (new_centroid_1 - centroid_1) * (new_centroid_1 - centroid_1);
    dist += (new_centroid_2 - centroid_2) * (new_centroid_2 - centroid_2);
    dist += sqrt((float)dist);

    atomicAdd(sum_loss, dist);
}

__global__ void KmeansWriteResult(unsigned char* image_src, unsigned char* image_result, \
            unsigned char* centroid, unsigned width, unsigned height, unsigned channels) {

    const int idx_x = blockIdx.x * blockDim.x + threadIdx.x;
    const int idx_y = blockIdx.y * blockDim.y + threadIdx.y;
    const int thread_width = blockDim.x * gridDim.x;
    const int thread_height = blockDim.y * gridDim.y;

    int min_dist = INF;
    int dist = 0;
    int cluster_idx;

    int img_src0 = image_src[channels * (idx_x + idx_y * width) + 0];
    int img_src1 = image_src[channels * (idx_x + idx_y * width) + 1];
    int img_src2 = image_src[channels * (idx_x + idx_y * width) + 2];

    __shared__ unsigned char shared_centroid[num_cluster][3];

    for(int k = (threadIdx.x + threadIdx.y * blockDim.x); k < num_cluster; k += (blockDim.x * blockDim.y)) {
        shared_centroid[k][0] = centroid[channels * k + 0];
        shared_centroid[k][1] = centroid[channels * k + 1];
        shared_centroid[k][2] = centroid[channels * k + 2];
    }

   __syncthreads();

    if(!(bound_check(idx_x, 0, width) && bound_check(idx_y, 0, height))) {
        return;
    }

    for(int k = 0; k < num_cluster; k++) {
        // calculate l2 norm
        dist = 0;
        dist += (img_src0 - shared_centroid[k][0]) * (img_src0 - shared_centroid[k][0]);
        dist += (img_src1 - shared_centroid[k][1]) * (img_src1 - shared_centroid[k][1]);
        dist += (img_src2 - shared_centroid[k][2]) * (img_src2 - shared_centroid[k][2]);
        dist = sqrt((float)dist);
        if(dist < min_dist) {
            min_dist = dist;
            cluster_idx = k;
        }
    }

    image_result[channels * (idx_x + idx_y * width) + 0] = shared_centroid[cluster_idx][0];
    image_result[channels * (idx_x + idx_y * width) + 1] = shared_centroid[cluster_idx][1];
    image_result[channels * (idx_x + idx_y * width) + 2] = shared_centroid[cluster_idx][2];
}

void kmeans(unsigned height, unsigned width, unsigned channels) {

    unsigned char *centroid = (unsigned char*) malloc(channels * num_cluster * sizeof(char));

    //unsigned char val[3];
    int dist, min_dist, idx, sum_loss;

    int *device_pt_cluster;
    int *device_sum_dist;
    int *device_num_pt_cluster;
    int *device_sum_loss;
    unsigned char *device_centroid, *device_new_centroid;

    hipMalloc(&device_sum_loss, 1 * sizeof(int));
    hipMalloc(&device_pt_cluster, height * width * sizeof(int));
    hipMalloc(&device_sum_dist, channels * num_cluster * sizeof(int));
    hipMalloc(&device_num_pt_cluster, num_cluster * sizeof(int));
    hipMalloc(&device_centroid, channels * num_cluster * sizeof(unsigned char));
    hipMalloc(&device_new_centroid, channels * num_cluster * sizeof(unsigned char));

    dim3 num_blocks(width / BlockFactor + 1, height / BlockFactor + 1);
    dim3 num_threads(BlockFactor, BlockFactor);

    // get random center
    for(int i = 0; i < num_cluster; i++) {
        int idx_i = rand() % width;
        int idx_j = rand() % height;
        centroid[channels * i + 0] = image_src[channels * (idx_i + idx_j * width) + 0];
        centroid[channels * i + 1] = image_src[channels * (idx_i + idx_j * width) + 1];
        centroid[channels * i + 2] = image_src[channels * (idx_i + idx_j * width) + 2];
    }

    hipMemcpy(device_centroid, centroid, channels * num_cluster * sizeof(unsigned char), hipMemcpyHostToDevice);
    hipMemcpy(device_image_src, image_src, channels * height * width * sizeof(unsigned char), hipMemcpyHostToDevice);

    while (1)
    {
        // Clustering All of the Pixels in image
        KmeansAssignCluster <<<num_blocks, num_threads>>> ( \
            device_image_src, device_centroid, device_num_pt_cluster, \
            device_sum_dist, height, width, channels \
        );

        // Clear the Value
        hipMemset(device_num_pt_cluster, 0, num_cluster * sizeof(int));
        hipMemset(device_sum_dist, 0, channels * num_cluster * sizeof(int));

        // Update Center & calculate the sum of difference between old center and new center & Store the Centroid Value
        KmeansCalLoss <<<1, num_cluster>>> (device_new_centroid, device_centroid, device_sum_loss, device_sum_dist, device_num_pt_cluster, channels);

        hipMemcpy(&sum_loss, device_sum_loss, 1 * sizeof(int), hipMemcpyDeviceToHost);

        // if the sum < threshold, stop the iteraton
        if(sum_loss < num_cluster * 4.1) {
            break;
        }
    }

    KmeansWriteResult <<<num_blocks, num_threads>>> ( \
        device_image_src, device_image_result, \
        device_centroid, width, height, channels \
    );

    hipMemcpy(image_result, device_image_result, height * width * channels * sizeof(unsigned char), hipMemcpyDeviceToHost);
}

int main(int argc, char** argv) {

    srand(time(0));

    assert(argc == 3);
    unsigned height, width, channels;
    image_src = NULL;
    
    read_png(argv[1], &image_src, &height, &width, &channels);

    image_result = (unsigned char*) malloc(height * width * channels * sizeof(unsigned char));
    
    hipHostRegister(image_src, height * width * channels * sizeof(unsigned char), hipHostRegisterDefault);

    hipMalloc(&device_image_src, height * width * channels * sizeof(unsigned char));
    hipMalloc(&device_image_result, height * width * channels * sizeof(unsigned char));

    kmeans(height, width, channels);

    write_png(argv[2], image_result, height, width, channels);

    free(image_src);
    free(image_result);
    hipFree(device_image_src);
    hipFree(device_image_result);

    return 0;
}
